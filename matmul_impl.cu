#include "hip/hip_runtime.h"
//
// Created by Pujun Lun on 2019-02-02.
//

#include "matmul_impl.h"

#include "macro.h"

namespace Math {
    static const uint TILE_WIDTH = 32;

    MatrixMultiplier::MatrixMultiplier(const Matrix &a, const Matrix &b, const Matrix &c,
            const Matrix *ref)
            : mC{c}, pRef{ref} {
        if (Matrix::verbose) {
            cout << "Extracting matrices data" << endl;
            cout << "A: " << a << endl;
            cout << "B: " << b << endl;
            cout << "C: " << c << endl;
        }

        m = a.rows() == c.rows() ?
            (uint)a.rows() : DEBUG_INFO("First dimension not match")
        n = b.cols() == c.cols() ?
            (uint)b.cols() : DEBUG_INFO("Second dimension not match")
        k = a.cols() == b.rows() ?
            (uint)a.cols() : DEBUG_INFO("Third dimension not match")

        sA = m * k * sizeof(float);
        sB = k * n * sizeof(float);
        sC = m * n * sizeof(float);

        hipMalloc(&pA, sA);
        hipMemcpy(pA, a.data(), sA, hipMemcpyHostToDevice);
        hipMalloc(&pB, sB);
        hipMemcpy(pB, b.data(), sB, hipMemcpyHostToDevice);
        hipMalloc(&pC, sC);
    }

    bool MatrixMultiplier::predicate() const {
        if (!pRef)
            DEBUG_INFO("No reference matrix")

        return mC == *pRef;
    }

    void MatrixMultiplier::transfer() const {
        hipMemcpy(mC.data(), pC, sC, hipMemcpyDeviceToHost);
    }

    void MatrixMultiplier::clear() const {
        mC.clear();
        hipMemset(pC, 0, sC);
    }

    MatrixMultiplier::~MatrixMultiplier() {
        if (Matrix::verbose)
            cout << "De-allocating matrices on device" << endl;

        hipFree(pA);
        hipFree(pB);
        hipFree(pC);
    }

    __global__ void matMulKernelA(const float *A, const float *B, float *C,
                                  const uint m, const uint n, const uint k) {
        uint row = blockIdx.y * TILE_WIDTH + threadIdx.y;
        uint col = blockIdx.x * TILE_WIDTH + threadIdx.x;

        float sum = 0.0f;
        for (uint i = 0; i < k; ++i)
            sum += A[row * k + i] * B[i * n + col];

        C[row * n + col] = sum;
    }

    __global__ void matMulKernelB(const float *A, const float *B, float *C,
                                  const uint m, const uint n, const uint k) {
        __shared__ float aD[TILE_WIDTH][TILE_WIDTH];
        __shared__ float bD[TILE_WIDTH][TILE_WIDTH];

        uint tx = threadIdx.x, ty = threadIdx.y;
        uint row = blockIdx.y * TILE_WIDTH + ty;
        uint col = blockIdx.x * TILE_WIDTH + tx;

        float sum = 0.0f;
        for (int i = 0; i < k / TILE_WIDTH; ++i) {
            aD[ty][tx] = A[row * k + (i * TILE_WIDTH + tx)];
            bD[ty][tx] = B[(i * TILE_WIDTH + ty) * n + col];
            __syncthreads();

            for (uint j = 0; j < TILE_WIDTH; ++j)
                sum += aD[ty][j] * bD[j][tx];
            __syncthreads();
        }
        C[row * n + col] = sum;
    }

    void matMulA(const MatrixMultiplier &multiplier) {
        if (multiplier.m % TILE_WIDTH != 0 || multiplier.n % TILE_WIDTH != 0)
            DEBUG_INFO("Dimension not supported")

        dim3 dimBlock{TILE_WIDTH, TILE_WIDTH};
        dim3 dimGrid{multiplier.n / TILE_WIDTH, multiplier.m / TILE_WIDTH};
        matMulKernelA<<<dimGrid, dimBlock>>>(
                multiplier.pA, multiplier.pB, multiplier.pC,
                multiplier.m, multiplier.n, multiplier.k);
    }

    void matMulB(const MatrixMultiplier &multiplier) {
        if (multiplier.m % TILE_WIDTH != 0 || multiplier.n % TILE_WIDTH != 0)
            DEBUG_INFO("Dimension not supported")

        dim3 dimBlock{TILE_WIDTH, TILE_WIDTH};
        dim3 dimGrid{multiplier.n / TILE_WIDTH, multiplier.m / TILE_WIDTH};
        matMulKernelB<<<dimGrid, dimBlock>>>(
                multiplier.pA, multiplier.pB, multiplier.pC,
                multiplier.m, multiplier.n, multiplier.k);
    }
}
