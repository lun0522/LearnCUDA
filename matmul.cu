#include "hip/hip_runtime.h"
//
// Created by Pujun Lun on 2019-01-25.
//

#include "matmul.h"

#include <chrono>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <>
#include <string>

#include "macro.h"

namespace Math {
    struct MatrixMultiplier;
    using KernelFunc = void (const MatrixMultiplier&);

    static const uint REPEAT_TIMES = 10;
    static const uint TILE_WIDTH = 32;

    struct MatrixMultiplier {
        uint m, n, k;
        uint sA, sB, sC;
        float *pA, *pB, *pC;
        const Matrix &mC, *pRef;

        MatrixMultiplier(const Matrix& a, const Matrix& b, const Matrix& c, const Matrix* ref)
            : mC {c}, pRef {ref} {
            if (Matrix::verbose) {
                cout << "Extracting matrices data" << endl;
                cout << "A: " << a << endl;
                cout << "B: " << b << endl;
                cout << "C: " << c << endl;
            }

            m = a.rows() == c.rows() ?
                (uint)a.rows() : DEBUG_INFO("First dimension not match")
            n = b.cols() == c.cols() ?
                (uint)b.cols() : DEBUG_INFO("Second dimension not match")
            k = a.cols() == b.rows() ?
                (uint)a.cols() : DEBUG_INFO("Third dimension not match")

            sA = m * k * sizeof(float);
            sB = k * n * sizeof(float);
            sC = m * n * sizeof(float);

            hipMalloc(&pA, sA);
            hipMemcpy(pA, a.data(), sA, hipMemcpyHostToDevice);
            hipMalloc(&pB, sB);
            hipMemcpy(pB, b.data(), sB, hipMemcpyHostToDevice);
            hipMalloc(&pC, sC);
        }

        bool predicate() const {
            if (!pRef) DEBUG_INFO("No reference matrix")
            return mC == *pRef;
        }

        void transfer() const {
            hipMemcpy(mC.data(), pC, sC, hipMemcpyDeviceToHost);
        }

        void clear() const {
            mC.clear();
            hipMemset(pC, 0, sC);
        }

        ~MatrixMultiplier() {
            if (Matrix::verbose)
                cout << "De-allocating matrices on device" << endl;

            hipFree(pA);
            hipFree(pB);
            hipFree(pC);
        }
    };

    __global__ void matMulKernelA(const float* A, const float* B, float* C,
                                  const uint m, const uint n, const uint k) {
        uint row = blockIdx.y * TILE_WIDTH + threadIdx.y;
        uint col = blockIdx.x * TILE_WIDTH + threadIdx.x;

        float sum = 0;
        for (uint i = 0; i < k; ++i)
            sum += A[row * k + i] * B[i * n + col];

        C[row * n + col] = sum;
    }

    void matMulA(const MatrixMultiplier& multiplier) {
        if (multiplier.m % TILE_WIDTH != 0 || multiplier.n % TILE_WIDTH != 0)
            DEBUG_INFO("Dimension not supported")

        dim3 dimBlock {TILE_WIDTH, TILE_WIDTH};
        dim3 dimGrid {multiplier.n / TILE_WIDTH, multiplier.m / TILE_WIDTH};
        matMulKernelA<<<dimGrid, dimBlock>>>(multiplier.pA, multiplier.pB, multiplier.pC,
                                             multiplier.m, multiplier.n, multiplier.k);
    }

    void verifyKernel(bool run, string name, KernelFunc func, const MatrixMultiplier& multiplier) {
        if (run) {
            func(multiplier);
            multiplier.transfer();
            if (!multiplier.predicate())
                DEBUG_INFO("Wrong result by kernel " + name)
            multiplier.clear();
        }
    }

    void repeatWithTimer(bool run, string name, KernelFunc func, const MatrixMultiplier& multiplier) {
        if (run) {
            using chrono::steady_clock;
            steady_clock::time_point begin = steady_clock::now();

            for (uint i = 0; i < REPEAT_TIMES; ++i) func(multiplier);
            hipDeviceSynchronize();

            steady_clock::time_point end = steady_clock::now();
            auto time = chrono::duration_cast<chrono::microseconds>(end - begin).count();
            cout << name << ": " << time / 1000.0 / REPEAT_TIMES << "ms" << std::endl;
        }
    }

    void testMatMul(const Matrix& a, const Matrix& b, MatMulAlgo algo) {
        if (Matrix::verbose)
            cout << "Multiplying " << a << " and " << b << endl;

        /* verify algorithms */
        if (Matrix::verbose)
            cout << endl << "Verifying correctness of algorithms" << endl;

        Matrix ref = blasMatMul(a, b);
        Matrix c {a.rows(), b.cols(), Matrix::Mode::undefined};
        MatrixMultiplier multiplier {a, b, c, &ref};

        verifyKernel(algo & MatMulAlgoA, "A", matMulA, multiplier);

        /* record elapsed time */
        if (Matrix::verbose)
            cout << endl << "Reporting elapsed time of algorithms" << endl;

        repeatWithTimer(algo & MatMulAlgoA, "A", matMulA, multiplier);
    }

    Matrix blasMatMul(const Matrix& a, const Matrix& b) {
        if (Matrix::verbose)
            cout << "Matrix multiply via cuBLAS" << endl;

        Matrix c {a.rows(), b.cols(), Matrix::Mode::undefined};
        MatrixMultiplier multiplier {a, b, c, nullptr};

        int lda = (int)multiplier.m, ldb = (int)multiplier.k, ldc = (int)multiplier.m;
        float alpha = 1.0f, beta = 0.0f;

        hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, multiplier.m, multiplier.n, multiplier.k,
                    &alpha, multiplier.pA, lda, multiplier.pB, ldb, &beta, multiplier.pC, ldc);
        hipblasDestroy(handle);

        multiplier.transfer();
        return c;
    }
}
